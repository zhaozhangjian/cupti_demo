#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_texture_types.h"
#include <hip/hip_runtime.h>
#include <cupti.h>

#define CUPTI_CALL(call)                                                   \
  do {                                                                     \
    CUptiResult _status = call;                                            \
    if (_status != CUPTI_SUCCESS) {                                        \
      const char *errstr;                                                  \
      cuptiGetResultString(_status, &errstr);                              \
      fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n", \
        __FILE__, __LINE__, #call, errstr);                                \
      exit(-1);                                                            \
    }                                                                      \
  } while (0)

static CUpti_SubscriberHandle cuptiSubscriber;

void cuptiSubscriberCallback(
  void *userdata,
  CUpti_CallbackDomain domain,
  CUpti_CallbackId cb_id,
  const CUpti_CallbackData *cb_info) {
  const char* apiName;
  cuptiGetCallbackName(domain, cb_id, &apiName);
  printf("callback: %d %d %d %p %s\n", domain, cb_id, cb_info->correlationId, cb_info, apiName);
}

void initTrace() {
  // Subscribe callbacks
  CUPTI_CALL(cuptiSubscribe(&cuptiSubscriber, (CUpti_CallbackFunc) cuptiSubscriberCallback, (void *) NULL));
  CUPTI_CALL(cuptiEnableDomain(1, cuptiSubscriber, CUPTI_CB_DOMAIN_DRIVER_API));
  CUPTI_CALL(cuptiEnableDomain(1, cuptiSubscriber, CUPTI_CB_DOMAIN_RUNTIME_API));
}

void finiTrace() {
  CUPTI_CALL(cuptiActivityFlushAll(CUPTI_ACTIVITY_FLAG_FLUSH_FORCED));
  CUPTI_CALL(cuptiUnsubscribe(cuptiSubscriber));
}

// Texture reference for 1D float texture
texture<float, 1, hipReadModeElementType> texRefUniqueName;

// Kernel that uses texture memory
__global__ void textureKernel(float* output, int width) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < width) {
      // Read from texture memory
      output[x] = tex1Dfetch(texRefUniqueName, x);
    }
}
// Kernel that NOT uses texture memory
__global__ void noTextureKernel(float* output, int width) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < width) {
      // Read from memory
      output[x] = output[x] + (float)x;
    }
}

int main() {
  const int N = 64;
  const textureReference* texref = nullptr;
  float* h_data = (float*)malloc(N * sizeof(float));
  float* h_output = (float*)malloc(N * sizeof(float));
  
  // Initialize host data
  for (int i = 0; i < N; i++) {
    h_data[i] = (float)i;
  }

  initTrace();

  // Allocate device memory
  float* d_data;
  float* d_output;
  hipMalloc((void**)&d_data, N * sizeof(float));
  hipMalloc((void**)&d_output, N * sizeof(float));
  
  // Copy data to device
  hipMemcpy(d_data, h_data, N * sizeof(float), hipMemcpyHostToDevice);

  // Bind texture to device memory
  hipBindTexture(NULL, texRefUniqueName, d_data, N * sizeof(float));
  hipGetTextureReference((const textureReference**)&texref, (void*)&texRefUniqueName);
  
  // Launch kernel
  dim3 blockDim(16);
  dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
  textureKernel<<<gridDim, blockDim>>>(d_output, N);
  
  noTextureKernel<<<gridDim, blockDim>>>(d_output, N);
  
  // Copy results back to host
  hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);
  
  // Verify results
  bool success = true;
  for (int i = 0; i < N; i++) {
    float golden = h_data[i] * 2.0f;
    if (h_output[i] != golden) {
      printf("Error at %d: %f != %f\n", i, h_output[i], golden);
      success = false;
    }
  }

  if (success) {
    printf("Texture memory test passed!\n");
  }

  // Cleanup
  hipUnbindTexture(texRefUniqueName);
  hipFree(d_data);
  hipFree(d_output);
  free(h_data);
  free(h_output);

  finiTrace();

  return 0;
}
